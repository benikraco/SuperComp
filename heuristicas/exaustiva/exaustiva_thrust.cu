#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/extrema.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <math.h>
using namespace std;

// Structure for movies
struct Movie {
    int start, end, category, index;
};

bool compareMovies(const Movie &a, const Movie &b) {
    if (a.end == b.end) {
        return a.start < b.start;
    }
    return a.end < b.end;
}

// Transformer to perform the calculation on the GPU
struct Transformer {
    int total_movies;
    int total_categories;
    int *max_movies_per_category;
    Movie *movie_list;

    Transformer(int _total_movies, int _total_categories, int* _max_movies_per_category, Movie *_movie_list) : 
        total_movies(_total_movies), total_categories(_total_categories), max_movies_per_category(_max_movies_per_category), movie_list(_movie_list) {};

    __device__ __host__
    thrust::tuple<int, unsigned long long> operator() (unsigned long long combination) {
        int movie_counter = 0;
        int count_per_category[10];
        bool time_slots[24];
        bool valid, finished;

        for (int i = 0; i < total_categories; i++) count_per_category[i] = 0;
        for (int i = 0; i < 24; i++) time_slots[i] = false;

        for (int i = 0; i < total_movies; i++) {
            valid = true;
            finished = true;
            unsigned long long comparison = pow(2, i);
            if ((combination & comparison) && (count_per_category[movie_list[i].category-1] < max_movies_per_category[movie_list[i].category-1])) {
                for (int j = movie_list[i].start; j < (movie_list[i].end > movie_list[i].start ? movie_list[i].end : 24); j++) {
                    if (time_slots[j]) {
                        valid = false;
                        break;
                    }
                    time_slots[j] = true;
                }
                if (valid) {
                    count_per_category[movie_list[i].category-1]++;
                    movie_counter++;
                    for (int j = 0; j < 10; j++) {
                        if (count_per_category[j] != max_movies_per_category[j]) {
                            finished = false;
                            break;
                        }
                    }
                    if (movie_counter >= 24 || finished) break;
                }
            }
        }
        return thrust::make_tuple(movie_counter, combination);
    }
};

// Custom comparison function to find the optimal combination
struct Comparator {
    __device__ __host__
    bool operator() (const thrust::tuple<int, unsigned long long>& a, const thrust::tuple<int, unsigned long long>& b) const {
        return thrust::get<0>(a) < thrust::get<0>(b);
    }
};


// Read movie information from a file
vector<Movie> readMovies(string file_name, int &N, int &M, vector<int> &max_movies_per_category) {
    ifstream input_file(file_name);

    input_file >> N >> M;
    max_movies_per_category.resize(M);
    for (int i = 0; i < M; i++) input_file >> max_movies_per_category[i];

    vector<Movie> movies(N);
    for (int i = 0; i < N; i++) {
        input_file >> movies[i].start >> movies[i].end >> movies[i].category;
        movies[i].index = i;
        if (movies[i].end < movies[i].start) movies[i].end += 24;
    }

    input_file.close();
    return movies;
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Usage: " << argv[0] << " <input_file>" << endl;
        return 1;
    }

    string input_file = argv[1];
    int number_of_movies, number_of_categories;
    vector<int> max_movies_per_category;

    // Reading movies from the file
    vector<Movie> movies = readMovies(input_file, number_of_movies, number_of_categories, max_movies_per_category);

    // Sorting the movies based on their end and start times
    sort(movies.begin(), movies.end(), compareMovies);

    thrust::device_vector<unsigned long long> possibilities(pow(2, number_of_movies));
    thrust::sequence(possibilities.begin(), possibilities.end());

    thrust::device_vector<int> max_categories_dev = max_movies_per_category;
    thrust::device_vector<Movie> movies_dev = movies;

    thrust::device_vector<thrust::tuple<int, unsigned long long>> final_output(pow(2, number_of_movies));

    auto start_calculation = chrono::high_resolution_clock::now();
    thrust::transform(
        possibilities.begin(),
        possibilities.end(),
        final_output.begin(),
        Transformer(number_of_movies, number_of_categories, thrust::raw_pointer_cast(max_categories_dev.data()), thrust::raw_pointer_cast(movies_dev.data()))
    );

    auto max_iter = thrust::max_element(final_output.begin(), final_output.end(), Comparator());
    auto end_calculation = chrono::high_resolution_clock::now();
    auto calculation_duration = chrono::duration_cast<chrono::microseconds>(end_calculation - start_calculation);

    thrust::host_vector<thrust::tuple<int, unsigned long long>> final_output_host = final_output;

    int max_movies = thrust::get<0>(final_output_host[max_iter - final_output.begin()]);
    unsigned long long max_combination = thrust::get<1>(final_output_host[max_iter - final_output.begin()]);

    cout << "Movies watched: " << max_movies << endl;
    unsigned long long comparison;
    for (int i = 0; i < number_of_movies; i++) {
        comparison = pow(2, i);
        if (max_combination & comparison) {
            cout << "Movie start: " << movies[i].start << ", end: " << (movies[i].end > movies[i].start ? movies[i].end : movies[i].end - 24) << ", category: " << movies[i].category << endl;
        }
    }

    cout << "Time elapsed during the exhaustive search algorithm: " << calculation_duration.count() << " microseconds" << endl;

    return 0;
}
